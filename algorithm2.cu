#include "hip/hip_runtime.h"

/*
 *
 * Copyright (C) 2020-  Yewang Chen<ywchen@hqu.edu.cn;nalandoo@gmail.com>
 * License: GPL v1
 * This software may be modified and distributed under the terms
 * of license.
 *
 */

#include "def.h"


 /*
 *
 * Gauss���棬ʹ��O(n^2)��ʱ�临�Ӷȣ������������̶߳���ͬ���Ĺ���
 **/
static __global__
void Gauss_Jordan_Inverse(__DATA_TYPE* mat_tmp, int size, int dy) {

	int idx = threadIdx.x;
	int idy = threadIdx.y;
	int bx = blockIdx.x;

	int dis = idy * size * size; //Compute the distance between idy-th matrix and the address of mat_tmp.

	if (idx >= size || idy >= dy) return;

	mat_tmp += bx * size * size * dy;
	extern __shared__ __DATA_TYPE mat[]; //using static shared memory 48 KB
	Matrix_copy_glob2shr(mat_tmp + idx * size + dis, mat + idx * size + dis, size);
	__syncthreads();

	int i, j, k;
	__DATA_TYPE c;
	for (k = 0; k < size; k++) {
		//1.m(k,k) = 1/m(k,k)
		mat[k * size + k + dis] = 1.0 / mat[k * size + k + dis];
		c = mat[k * size + k + dis];

		//2.m(i,k) = -m(k,k) * m(i,k), i!=k
		if (idx != k) mat[idx * size + k + dis] *= -1 * c;

		__syncthreads();
		//3.m(i,j) = m(i,j) + m(i,k) * m(k,j), i,j != k
		for (i = 0; i < k; i++) {
			if (idx != k) mat[i * size + idx + dis] += mat[i * size + k + dis] * mat[k * size + idx + dis];
		}
		for (i = k + 1; i < size; i++) {
			if (idx != k) mat[i * size + idx + dis] += mat[i * size + k + dis] * mat[k * size + idx + dis];
		}

		//4.m(k,j) = m(k,k) * m(k,j), j != k
		if (idx != k)  mat[k * size + idx + dis] *= c;
		__syncthreads();
	}

	Matrix_copy_shr2glob(mat + idx * size + dis, mat_tmp + idx * size + dis, size);
}


//static __global__
//void Gauss_Jordan_Inverse(__DATA_TYPE* mat_tmp, int size, int dy) {
//
//	register int idx = threadIdx.x%size;
//	register int idy = threadIdx.x/size;
//	register int bx = blockIdx.x;
//
//	register int dis = idy * size * size; //Compute the distance between idy-th matrix and the address of mat_tmp.
//
//	if (idx >= size || idy >= dy) return;
//
//	mat_tmp += bx * size * size * dy;
//	extern __shared__ __DATA_TYPE mat[]; //using static shared memory 48 KB
//	Matrix_copy_glob2shr(mat_tmp + idx * size + dis, mat + idx * size + dis, size);
//	__syncthreads();
//
//	int i, j, k;
//	__DATA_TYPE c;
//	register int ksize = dis;
//	for (k = 0; k < size; k++) {
//		//1.m(k,k) = 1/m(k,k)
//		register int mid = ksize + k;
//		mat[mid] = 1.0 / mat[mid];
//		c = mat[mid];
//
//		//2.m(i,k) = -m(k,k) * m(i,k), i!=k
//		if (idx != k) mat[idx * size + k + dis] *= -1 * c;
//
//		__syncthreads();
//		//3.m(i,j) = m(i,j) + m(i,k) * m(k,j), i,j != k
//		register int tem = dis;
//		for (i = 0; i < size; i++) {
//			if (idx != k ||i!=k) mat[tem + idx] += mat[tem + k] * mat[ksize + idx];
//			tem += size;
//		}
//
//		//4.m(k,j) = m(k,k) * m(k,j), j != k
//		if (idx != k)  mat[ksize + idx] *= c;
//		__syncthreads();
//		ksize += size;
//	}
//
//	Matrix_copy_shr2glob(mat + idx * size + dis, mat_tmp + idx * size + dis, size);
//}

static
int single_sm_inverse_gauss_gpu(__DATA_TYPE* out, int size, int my_np) {

	int deviceSno = 0;
	hipSetDevice(deviceSno);

	hipEvent_t start, stop;
	hipError_t cudaerror = hipSuccess;

	DeviceInfo d;
	GetDeviceInfo(d);
	Device* ds = d.device;
	while (ds) {
		if (ds->index == deviceSno)
			break;
		else ds = ds->next;
	}
	int maxbytes = ds->sharedMemPerBlockOptin; // 65535 byte = 64 KB
	hipFuncSetAttribute(reinterpret_cast<const void*>(Gauss_Jordan_Inverse), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
	int dy = min(floor(ds->sharedMemPerBlockOptin * 1.0 / (size * size * sizeof(__DATA_TYPE))), my_np);
	dy = min(1024 / size, dy); //dy->[1,1024]
	int matMaxSize = floor(sqrt(maxbytes * 1.0 / sizeof(__DATA_TYPE)));

	//Output some necessary infomation for remind you!
	int remain = my_np % dy;
	printf("%d matrix inverse works per block,MatMaxSize:%d\n", dy, matMaxSize);
	if (remain) {
		printf("Notice:There have %d last matrixes will not be inversed,You would better set my_np to multiples of %d!!\n", remain, dy);
	}

	float elapse_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	__DATA_TYPE* d_out;

	//hipEventRecord(start, 0);
	gpuErrchk(hipMalloc((void**)&d_out, my_np * size * size * sizeof(__DATA_TYPE)));
	gpuErrchk(hipMemcpy(d_out, out, my_np * size * size * sizeof(__DATA_TYPE), hipMemcpyHostToDevice));

	dim3 blocks(my_np / dy), threads(size, dy);
	hipEventRecord(start, 0);
	Gauss_Jordan_Inverse << <blocks, threads, maxbytes >> > (d_out, size, dy);
	hipEventRecord(stop, 0);

	gpuErrchk(hipMemcpy(out, d_out, my_np * size * size * sizeof(__DATA_TYPE), hipMemcpyDeviceToHost));
	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record֮ǰ������

	hipEventElapsedTime(&elapse_time, start, stop);    //����ʱ���
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_out);

	printf("ִ��ʱ�䣺%f(ms)\n", elapse_time);
	cudaerror = hipGetLastError();

	free_device_list(d.device);
	//record time
	writeGPUResults(elapse_time);

	if (cudaerror != hipSuccess) {
		cudaCheck(cudaerror);
		return 0;
	}
	return 1;
}

static
int more_sm_inverse_gauss_gpu(__DATA_TYPE* out, int size, int my_np) {

	int deviceSno = 0;
	hipSetDevice(deviceSno);

	hipEvent_t start, stop;
	hipError_t cudaerror = hipSuccess;
	float elapse_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	DeviceInfo d;
	GetDeviceInfo(d);
	Device* ds = d.device;
	while (ds) {
		if (ds->index == deviceSno)
			break;
		else ds = ds->next;
	}
	int maxbytes = ds->sharedMemPerBlockOptin; // 65535 byte = 64 KB
	hipFuncSetAttribute(reinterpret_cast<const void*>(Gauss_Jordan_Inverse), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

	int sm = ds->SMCount;
	int tasksPerStream = my_np / sm;
	int dis = tasksPerStream * size * size;
	//Compute number of works every block.
	/*int remain = my_np % sm;
	if (remain) {
		printf("MSM Notice:there have %d last matrix(es) won't be inversed,becasuse be ignored. You would be better set my_np to mutilples of %d. \n\n",remain, sm);
	}*/

	int dy = min(floor(ds->sharedMemPerBlockOptin * 1.0 / (size * size * sizeof(__DATA_TYPE))), tasksPerStream);
	dy = min(1024 / size, dy); //dy->[1,1024]
	//int matMaxSize = floor(sqrt(maxbytes * 1.0 / sizeof(__DATA_TYPE)));


	__DATA_TYPE* d_out;

	hipStream_t* stream = new hipStream_t[sm];
	for (int i = 0; i < sm; i++) {
		gpuErrchk(hipStreamCreate(&stream[i]));
	}

	/*printf("%d matrix inverse works per block,MatMaxSize:%d\n", dy, matMaxSize);
	remain = tasksPerStream % dy;
	printf("remain:%d\n", remain);
	if (remain) {
		printf("MulSm Notice:There have %d last matrixes will not be inversed,it will happen every %d matrixes,You would better set my_np to multiples of %d!!\n", remain,tasksPerStream, sm*dy);
	}*/

	dim3 blocks(tasksPerStream / dy), threads(size,dy);


	gpuErrchk(hipMalloc((void**)&d_out, my_np * size * size * sizeof(__DATA_TYPE)));

	for (int i = 0; i < sm; i++) {
		gpuErrchk(hipMemcpyAsync(d_out + i * dis, out + i * dis, dis * sizeof(__DATA_TYPE), hipMemcpyHostToDevice, stream[i]));
	}

	hipEventRecord(start, 0);
	for (int i = 0; i < sm; i++) {
		Gauss_Jordan_Inverse << <blocks, threads, maxbytes, stream[i] >> > (d_out + i * dis, size, dy);
	}
	hipEventRecord(stop, 0);

	for (int i = 0; i < sm; i++) {
		gpuErrchk(hipMemcpyAsync(out + i * dis, d_out + i * dis, dis * sizeof(__DATA_TYPE), hipMemcpyDeviceToHost, stream[i]));
	}

	cudaerror = hipGetLastError();

	for (int i = 0; i < sm; ++i) {
		gpuErrchk(hipStreamDestroy(stream[i]));
	}

	free_device_list(d.device);
	gpuErrchk(hipFree(d_out));


	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record֮ǰ������
	hipEventElapsedTime(&elapse_time, start, stop);    //����ʱ���
	hipEventDestroy(start);
	hipEventDestroy(stop);
	writeGPUResults(elapse_time);

	printf("ִ��ʱ�䣺%f(ms)\n", elapse_time);
	if (cudaerror != hipSuccess) {
		return 0;
	}
	return 1;


}

static
int get_devided_number_of_single_to_mul_sm(int size, int my_np) {
	int deviceSno = 0;
	DeviceInfo d;
	GetDeviceInfo(d);
	Device* ds = d.device;
	while (ds) {
		if (ds->index == deviceSno)
			break;
		else ds = ds->next;
	}
	int maxbytes = ds->sharedMemPerBlockOptin; // 65535 byte = 64 KB
	int dy = min(floor(ds->sharedMemPerBlockOptin * 1.0 / (size * size * sizeof(__DATA_TYPE))), my_np);
	dy = min(1024 / size, dy); //dy->[1,1024]
	free_device_list(d.device);
	return ceil(dy * 65536 * 1.0 / 35);
}


int my_algorithm2(__DATA_TYPE* out, int size, int my_np) {

	if (size > 90) {
		printf("ERROR!!! The method allow the size of matrix small than 90!!!\n");
		return 0;
	}
	/*int limit = get_devided_number_of_single_to_mul_sm(size,my_np);

	printf("Limit number of matrixes:%d\n", limit);*/

	if (my_np >= 1 && my_np < 1024) {
		return single_sm_inverse_gauss_gpu(out, size, my_np);
	}
	else {
		return more_sm_inverse_gauss_gpu(out, size, my_np);
	}
}