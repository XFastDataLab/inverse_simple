
#include "def.h"
#include "hipblas.h"

int my_gauss_inverse_gpu_by_cublas(float** d_in, int size, float** d_out, int my_np) {

	
	hipError_t cudaerror = hipSuccess;

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int* info, * pivo;
	
	hipMalloc((void**)&info, sizeof(int) * my_np);
	hipMalloc((void**)&pivo, sizeof(int) * size * my_np);


	float** gpuMat;
	hipMalloc((void**)&gpuMat, sizeof(float*) * my_np);
	hipMemcpy(gpuMat, d_in, sizeof(float*) * my_np, hipMemcpyHostToDevice);

	
	float** gpuInvMat;
	hipMalloc((void**)&gpuInvMat, sizeof(float*) * my_np);
	hipMemcpy(gpuInvMat, d_out, sizeof(float*) * my_np, hipMemcpyHostToDevice);

	
	hipblasSgetrfBatched(handle, size, gpuMat, size, pivo, info, my_np);

	hipDeviceSynchronize();
	hipblasSgetriBatched(handle, size, gpuMat, size, pivo, gpuInvMat, size, info, my_np);

	hipDeviceSynchronize();
	
	hipFree(info);
	hipFree(pivo);
	hipFree(gpuMat);
	hipFree(gpuInvMat);
	/*for (int i = 0; i < my_np; i++) {
		hipFree(resulthd[i]);
	}*/

	return 1;
}