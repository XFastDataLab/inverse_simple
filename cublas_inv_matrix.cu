
#include "def.h"
#include "hipblas.h"

int my_gauss_inverse_gpu_by_cublas(float** d_in, int size, float** d_out, int my_np) {

	
	hipError_t cudaerror = hipSuccess;

	hipblasHandle_t handle;
	hipEvent_t start, stop, start1,end1, start2,end2;
	float time_elapsed = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start1);
	hipEventCreate(&end1);
	hipEventCreate(&start2);
	hipEventCreate(&end2);


	

	hipblasCreate(&handle);
	int* info, * pivo;
	
	hipMalloc((void**)&info, sizeof(int) * my_np);
	hipMalloc((void**)&pivo, sizeof(int) * size * my_np);


	float** gpuMat;
	hipMalloc((void**)&gpuMat, sizeof(float*) * my_np);
	hipMemcpy(gpuMat, d_in, sizeof(float*) * my_np, hipMemcpyHostToDevice);

	float** resulthd = new float* [my_np];
	for (int i = 0; i < my_np; i++) {
		hipMalloc((void**)&resulthd[i], sizeof(float) * size * size);
	}
	
	float** gpuInvMat;
	hipMalloc((void**)&gpuInvMat, sizeof(float*) * my_np);
	hipMemcpy(gpuInvMat, resulthd, sizeof(float*) * my_np, hipMemcpyHostToDevice);

	tools_gpuAssert(hipEventRecord(start, 0));
	hipblasSgetrfBatched(handle, size, gpuMat, size, pivo, info, my_np);
	hipDeviceSynchronize();
	hipDeviceSynchronize();
	hipblasSgetriBatched(handle, size, gpuMat, size, pivo, gpuInvMat, size, info, my_np);
	hipDeviceSynchronize();
	hipDeviceSynchronize();

	tools_gpuAssert(hipEventRecord(stop, 0));


	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record֮ǰ������
	tools_gpuAssert(hipEventElapsedTime(&time_elapsed, start, stop));    //����ʱ���



	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(start1);
	hipEventDestroy(end1);
	hipEventDestroy(start2);
	hipEventDestroy(end2);


	printf("ִ��ʱ�䣺%f(ms)\n", time_elapsed);
	cout << "before" << endl;
	for (int i = 0; i < my_np; i++) {
		hipMemcpy(d_out[i], resulthd[i], sizeof(float) * size * size, hipMemcpyDeviceToHost);
	}
	cout << "memcpy" << endl;
	writeGPUResults(time_elapsed);

	hipFree(info);
	hipFree(pivo);
	hipFree(gpuMat);
	hipFree(gpuInvMat);

	for (int i = 0; i < my_np; i++) {
		hipFree(resulthd[i]);
	}

	hipblasDestroy(handle);
	cout << "hhh" << endl;
	return 1;
}