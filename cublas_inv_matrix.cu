
#include "def.h"
#include "hipblas.h"

int my_gauss_inverse_gpu_by_cublas(float** d_in, int size, float** d_out, int my_np) {

	hipEvent_t start, stop;
	float time_elapsed = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipError_t cudaerror = hipSuccess;

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int* info, * pivo;
	
	hipMalloc((void**)&info, sizeof(int) * my_np);
	hipMalloc((void**)&pivo, sizeof(int) * size * my_np);


	float** gpuMat;
	hipMalloc((void**)&gpuMat, sizeof(float*) * my_np);
	hipMemcpy(gpuMat, d_in, sizeof(float*) * my_np, hipMemcpyDeviceToDevice);

	
	float** gpuInvMat;
	hipMalloc((void**)&gpuInvMat, sizeof(float*) * my_np);
	hipMemcpy(gpuInvMat, d_out, sizeof(float*) * my_np, hipMemcpyDeviceToDevice);

	hipEventRecord(start, 0);
	hipblasSgetrfBatched(handle, size, gpuMat, size, pivo, info, my_np);

	hipDeviceSynchronize();
	hipblasSgetriBatched(handle, size, gpuMat, size, pivo, gpuInvMat, size, info, my_np);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	
	/*for (int i = 0; i < my_np; i++) {
		hipMemcpy(d_out[i], resulthd[i], sizeof(float) * size * size, hipMemcpyDeviceToHost);
	}*/

	

	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record֮ǰ������
	hipEventElapsedTime(&time_elapsed, start, stop);    //����ʱ���
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("ִ��ʱ�䣺%f(ms)\n", time_elapsed);

	writeGPUResults(time_elapsed);

	hipFree(info);
	hipFree(pivo);
	hipFree(gpuMat);
	hipFree(gpuInvMat);
	/*for (int i = 0; i < my_np; i++) {
		hipFree(resulthd[i]);
	}*/

	return 1;
}