
#include "def.h"
#include "hipblas.h"

int my_gauss_inverse_gpu_by_cublas(float** d_in, int size, float** d_out, int my_np) {

	
	hipError_t cudaerror = hipSuccess;

	hipblasHandle_t handle;
	hipEvent_t start, stop, start1,end1, start2,end2;
	float time_elapsed = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start1);
	hipEventCreate(&end1);
	hipEventCreate(&start2);
	hipEventCreate(&end2);


	

	hipblasCreate(&handle);
	int* info, * pivo;
	
	tools_gpuAssert(hipMalloc((void**)&info, sizeof(int) * my_np));
	tools_gpuAssert(hipMalloc((void**)&pivo, sizeof(int) * size * my_np));


	float*gpuMat, **A = (float**)malloc(my_np * sizeof(float*)), **A_d;
	tools_gpuAssert(hipMalloc((void**)&gpuMat, sizeof(float) * my_np * size * size));
	tools_gpuAssert(hipMalloc((void**)&A_d, sizeof(float*) * my_np));

	A[0] = gpuMat;
	for (int i = 1; i < my_np; i++) {
		A[i] = A[i - 1] + size * size;
	}

	tools_gpuAssert(hipMemcpy(A_d, A, my_np * sizeof(float*), hipMemcpyHostToDevice));
	
	for (int i = 0; i < my_np; i++) {
		tools_gpuAssert(hipMemcpy(gpuMat + (i * size * size), d_in[i], size * size * sizeof(float), hipMemcpyHostToDevice));
	}


	float** C = (float**)malloc(my_np * sizeof(float*));
	float** C_d, * C_dflat;

	hipMalloc(&C_d, my_np * sizeof(float*));
	hipMalloc(&C_dflat, size * size * my_np * sizeof(float));
	C[0] = C_dflat;
	for (int i = 1; i < my_np; i++) {
		C[i] = C[i - 1] + (size * size);
	}

	tools_gpuAssert(hipMemcpy(C_d, C, my_np * sizeof(float*), hipMemcpyHostToDevice));



	tools_gpuAssert(hipEventRecord(start, 0));
	hipblasSgetrfBatched(handle, size, A_d, size, pivo, info, my_np);
	hipDeviceSynchronize();
	hipDeviceSynchronize();
	hipblasSgetriBatched(handle, size, A_d, size, pivo, C_d, size, info, my_np);
	hipDeviceSynchronize();
	hipDeviceSynchronize();

	tools_gpuAssert(hipEventRecord(stop, 0));


	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record֮ǰ������
	tools_gpuAssert(hipEventElapsedTime(&time_elapsed, start, stop));    //����ʱ���



	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(start1);
	hipEventDestroy(end1);
	hipEventDestroy(start2);
	hipEventDestroy(end2);


	printf("ִ��ʱ�䣺%f(ms)\n", time_elapsed);
	cout << "before" << endl;
	for (int i = 0; i < my_np; i++) {
		hipMemcpy(d_out[i], C_dflat+(i*size*size), sizeof(float) * size * size, hipMemcpyDeviceToHost);
	}
	cout << "memcpy" << endl;
	writeGPUResults(time_elapsed);

	hipFree(info);
	hipFree(pivo);
	hipFree(gpuMat);


	free(A);
	hipFree(A_d);
	hipFree(C_d);
	hipFree(C_dflat);
	free(C);
	hipblasDestroy(handle);
	cout << "hhh" << endl;
	return 1;
}