
#include "def.h"
#include "hipblas.h"

int my_gauss_inverse_gpu_by_cublas(float** d_in, int size, float** d_out, int my_np) {

	
	hipError_t cudaerror = hipSuccess;

	hipblasHandle_t handle;
	hipEvent_t start, stop;
	float time_elapsed = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	tools_gpuAssert(hipEventRecord(start, 0));

	hipblasCreate(&handle);
	int* info, * pivo;
	
	hipMalloc((void**)&info, sizeof(int) * my_np);
	hipMalloc((void**)&pivo, sizeof(int) * size * my_np);


	float** gpuMat;
	hipMalloc((void**)&gpuMat, sizeof(float*) * my_np);
	hipMemcpy(gpuMat, d_in, sizeof(float*) * my_np, hipMemcpyHostToDevice);

	float** resulthd = new float* [my_np];
	for (int i = 0; i < my_np; i++) {
		hipMalloc((void**)&resulthd[i], sizeof(float) * size * size);
	}
	
	float** gpuInvMat;
	hipMalloc((void**)&gpuInvMat, sizeof(float*) * my_np);
	hipMemcpy(gpuInvMat, resulthd, sizeof(float*) * my_np, hipMemcpyHostToDevice);

	
	hipblasSgetrfBatched(handle, size, gpuMat, size, pivo, info, my_np);

	hipDeviceSynchronize();
	hipblasSgetriBatched(handle, size, gpuMat, size, pivo, gpuInvMat, size, info, my_np);

	hipDeviceSynchronize();

	tools_gpuAssert(hipEventRecord(stop, 0));
	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record֮ǰ������
	tools_gpuAssert(hipEventElapsedTime(&time_elapsed, start, stop));    //����ʱ���
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("ִ��ʱ�䣺%f(ms)\n", time_elapsed);

	for (int i = 0; i < my_np; i++) {
		hipMemcpy(d_out[i], resulthd[i], sizeof(float) * size * size, hipMemcpyDeviceToHost);
	}
	
	writeGPUResults(time_elapsed);

	hipFree(info);
	hipFree(pivo);
	hipFree(gpuMat);
	hipFree(gpuInvMat);

	for (int i = 0; i < my_np; i++) {
		hipFree(resulthd[i]);
	}

	hipblasDestroy(handle);

	return 1;
}